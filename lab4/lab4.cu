#include "hip/hip_runtime.h"
#include <wb.h>
#include <iostream>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
#define MASK_WIDTH 3
#define TILE_WIDTH 8


//@@ Define constant memory for device kernel here
__constant__ float Mc[MASK_WIDTH][MASK_WIDTH][MASK_WIDTH];

__global__ void conv3d(float *input, float *output, const int z_size,
                       const int y_size, const int x_size) {
  //@@ Insert kernel code here
  const int N_width = TILE_WIDTH + MASK_WIDTH - 1;
  __shared__ float N_ds[N_width][N_width][N_width];

  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  // printf("tidx= %d %d %d\n", tx, ty, tz);
  // printf("value= %d\n", threadIdx.x);

  int col_o = blockIdx.x * TILE_WIDTH + tx;
  int row_o = blockIdx.y * TILE_WIDTH + ty;
  int hei_o = blockIdx.z * TILE_WIDTH + tz;
  // printf("out= %d %d %d\n", row, col, hei);

  int col_i = col_o - MASK_WIDTH / 2;
  int row_i = row_o - MASK_WIDTH / 2;
  int hei_i = hei_o - MASK_WIDTH / 2;
  // printf("in= %d %d %d\n", row_i, col_i, hei_i);
  
  float Pvalue = 0.0f;

  if((col_i >= 0) && (col_i < x_size) &&
    (row_i >= 0) && (row_i < y_size) &&
    (hei_i >= 0) && (hei_i < z_size))
  {
    int i = hei_i * y_size * x_size + row_i * x_size + col_i;
    N_ds[tz][ty][tx] = input[i];
  }
  else
  {
    N_ds[tz][ty][tx] = 0.0f;
  }

  __syncthreads();


  if(tz < TILE_WIDTH && ty < TILE_WIDTH && tx < TILE_WIDTH)
  {
    for(int i = 0; i < MASK_WIDTH; i++)
    {
      for(int j = 0; j < MASK_WIDTH; j++)
      {
        for(int k = 0; k < MASK_WIDTH; k++)
        {
          Pvalue += Mc[i][j][k] * N_ds[i+tz][j+ty][k+tx];
        }
      }
    }
    if(col_o < x_size && row_o < y_size && hei_o < z_size)
    {
      // printf("pval: %.2f\n", Pvalue);
      output[hei_o*y_size*x_size + row_o*x_size + col_o] = Pvalue;
    }
  }
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  //@@ Initial deviceInput and deviceOutput here.
  float *deviceInput;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel =
      (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);

  //@@ Allocate GPU memory here
  // Recall that inputLength is 3 elements longer than the input data
  // because the first  three elements were the dimensions
  int size = (inputLength - 3) * sizeof(float);
  int sizeK = kernelLength * sizeof(float);

  hipMalloc((void **) &deviceInput, size);
  hipMalloc((void **) &deviceOutput, size);

  //@@ Copy input and kernel to GPU here
  // Recall that the first three elements of hostInput are dimensions and
  // do
  // not need to be copied to the gpu
  hipMemcpy(deviceInput, &hostInput[3], size, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(Mc), hostKernel, sizeK);

  //@@ Initialize grid and block dimensions here
  dim3 DimBlock(TILE_WIDTH+MASK_WIDTH-1, TILE_WIDTH+MASK_WIDTH-1, TILE_WIDTH+MASK_WIDTH-1);
  dim3 DimGrid(ceil(x_size / (1.0*TILE_WIDTH)), ceil(y_size / (1.0*TILE_WIDTH)), ceil(z_size / (1.0*TILE_WIDTH)));

  //@@ Launch the GPU kernel here
  conv3d<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, z_size, y_size, x_size);
  hipDeviceSynchronize();

  //@@ Copy the device memory back to the host here
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)
  hipMemcpy(&hostOutput[3], deviceOutput, size, hipMemcpyDeviceToHost);

  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);

  //@@ Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  // Free host memory
  free(hostInput);
  free(hostOutput);

  return 0;
}

