#include "hip/hip_runtime.h"
// LAB 1
#include <wb.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i<len)
    out[i] = in1[i] + in2[i];
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *in1_d, *in2_d, *out_d;
  // int size = inputLength * sizeof(float);

  args = wbArg_read(argc, argv);
  //@@ Importing data and creating memory on host
  hostInput1 =
      (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  wbLog(TRACE, "The input length is ", inputLength);

  //@@ Allocate GPU memory here
  hipMalloc((void **) &in1_d, inputLength * sizeof(float));
  hipMalloc((void **) &in2_d, inputLength * sizeof(float));
  hipMalloc((void **) &out_d, inputLength * sizeof(float));

  //@@ Copy memory to the GPU here
  hipMemcpy(in1_d, hostInput1, inputLength * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(in2_d, hostInput2, inputLength * sizeof(float), hipMemcpyHostToDevice);

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(ceil(inputLength/256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);

  //@@ Launch the GPU Kernel here to perform CUDA computation
  vecAdd<<<DimGrid, DimBlock>>>(in1_d, in2_d, out_d, inputLength);

  hipDeviceSynchronize();

  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, out_d, inputLength * sizeof(float), hipMemcpyDeviceToHost);

  //@@ Free the GPU memory here
  hipFree(in1_d);
  hipFree(in2_d);
  hipFree(out_d);


  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
